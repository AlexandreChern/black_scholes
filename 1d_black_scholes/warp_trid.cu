#include "hip/hip_runtime.h"
//
// This code tests the correct working of the routines in
// the trid.h header file which solve tridiagonal systems
// within a warp.
//

//
// standard header files
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//
// my tridiagonal header file
//

#include <trid.h>

//
// compute and print rms error
//

template <typename REAL>
__forceinline__ __device__ REAL rms_err(REAL err){
  err = err*err;
  for (uint i=16; i>=1; i=i/2) err += __shfl_xor(err,i);
  return sqrt(err/32.0);
}

template <typename REAL>
__forceinline__ __device__ REAL max_err(REAL err){
  err = fabs(err);
  for (uint i=16; i>=1; i=i/2) err = max(err,__shfl_xor(err,i));
  return err;
}

//
// test kernels
//

template <typename REAL>
__global__ void trid1_warp_test(REAL con) {

  REAL a, c, d, u;
  int  t = threadIdx.x%32;

  a = -0.5f;
  c = -0.5f;
  u =  2.0*t + 1.0f;

  if (t==0)  a = 0.0f;  // essential to zero these out;
  if (t==31) c = 0.0f;  // trid_warp needs them to be zero

  d = a*__shfl_up(u,1) + u + c*__shfl_down(u,1);

  d = trid1_warp(a,c,d);

  REAL err_rms = rms_err(d-u);
  REAL err_max = max_err(d-u);
  if(threadIdx.x==0)
    printf("rms err = %g, max err = %g \n",err_rms,err_max);
}


template <typename REAL>
__global__ void trid1_warp_new_test(REAL con) {

  REAL a, b, c, d, u;
  int  t = threadIdx.x%32;

  a = -0.5f;
  b =  1.0f;
  c = -0.5f;
  u =  2.0*t + 1.0f;

  if (t==0)  a = 0.0f;  // essential to zero these out;
  if (t==31) c = 0.0f;  // trid_warp needs them to be zero

  d = a*__shfl_up(u,1) + b*u + c*__shfl_down(u,1);

  d = trid1_warp_new(a,b,c,d);

  REAL err_rms = rms_err(d-u);
  REAL err_max = max_err(d-u);
  if(threadIdx.x==0)
    printf("rms err = %g, max err = %g \n",err_rms,err_max);
}


template <typename REAL>
__global__ void trid2_warp_test(REAL con) {

  REAL am, cm, dm, um, ap, cp, dp, up;
  int  t = threadIdx.x%32;

  am = -0.5f;
  cm = -0.5f;
  um =  2.0f*t + 1.0f;

  ap = -0.25f;
  cp = -0.25f;
  up =  2.0f*t + 2.0f;

  if (t==0)  am = 0.0f;  // essential to zero these out;
  if (t==31) cp = 0.0f;  // trid2_warp needs them to be zero

  dm = am*__shfl_up(up,1) + um + cm*up;
  dp = ap*um + up +  cp*__shfl_down(um,1);

  trid2_warp(am,cm,dm,ap,cp,dp);

  dm = dm - um;
  dp = dp - up;
  REAL err_rms = rms_err(sqrt(0.5f*(dm*dm+dp*dp)));
  REAL err_max = max_err(max(dm,dp));
  if(threadIdx.x==0)
    printf("rms err = %g, max err = %g \n",err_rms,err_max);
}



template <typename REAL>
__global__ void trid2_warp_s_test(REAL con) {

  __shared__ volatile REAL shared[32];

  REAL am, cm, dm, um, ap, cp, dp, up;
  int  t = threadIdx.x%32;

  am = -0.5f;
  cm = -0.5f;
  um =  2.0f*t + 1.0f;

  ap = -0.25f;
  cp = -0.25f;
  up =  2.0f*t + 2.0f;

  if (t==0)  am = 0.0f;  // essential to zero these out;
  if (t==31) cp = 0.0f;  // trid2_warp needs them to be zero

  dm = am*__shfl_up(up,1) + um + cm*up;
  dp = ap*um + up +  cp*__shfl_down(um,1);

  trid2_warp_s(am,cm,dm,ap,cp,dp,shared);

  dm = dm - um;
  dp = dp - up;
  REAL err_rms = rms_err(sqrt(0.5f*(dm*dm+dp*dp)));
  REAL err_max = max_err(max(dm,dp));
  if(threadIdx.x==0)
    printf("rms err = %g, max err = %g \n",err_rms,err_max);
}


//
// main code
//

int main(int argc, char **argv) {

  printf("\ntrid1_warp_test \n---------------\n");
  trid1_warp_test<<<1,32>>>(1.0f);  // single precision test
  hipDeviceSynchronize();
  trid1_warp_test<<<1,32>>>(1.0);   // double precision test
  hipDeviceSynchronize();

  printf("\ntrid1_warp_new_test \n---------------\n");
  trid1_warp_new_test<<<1,32>>>(1.0f);  // single precision test
  hipDeviceSynchronize();
  trid1_warp_new_test<<<1,32>>>(1.0);   // double precision test
  hipDeviceSynchronize();

  printf("\ntrid2_warp_test \n---------------\n");
  trid2_warp_test<<<1,32>>>(1.0f);  // single precision test
  hipDeviceSynchronize();
  trid2_warp_test<<<1,32>>>(1.0);   // double precision test
  hipDeviceSynchronize();

  printf("\ntrid2_warp_s_test \n-----------------\n");
  trid2_warp_s_test<<<1,32>>>(1.0f);  // single precision test
  hipDeviceSynchronize();
  trid2_warp_s_test<<<1,32>>>(1.0);   // double precision test
  hipDeviceSynchronize();

// CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
  return 0;
}
